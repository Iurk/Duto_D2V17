#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <iomanip>
#include <string>

#define _USE_MATH_DEFINES
#include <math.h>

#include <hip/hip_runtime.h>
#include <dirent.h>
#include <errno.h>
#include <sys/stat.h>

#include "paths.h"
#include "LBM.h"
#include "dados.h"

using namespace myGlobals;

// Input data
__constant__ unsigned int q, Nx_d, Ny_d;
__constant__ double rho0_d, u_max_d, nu_d, tau_d, mi_ar_d;

//Lattice Data
__constant__ double as_d, w0_d, wp_d, ws_d, wt_d, wq_d;
__device__ int *ex_d;
__device__ int *ey_d;

// Mesh data
__device__ bool *solid_d;

__device__ __forceinline__ size_t gpu_field0_index(unsigned int x, unsigned int y){
	return Nx_d*y + x;
}

__device__ __forceinline__ size_t gpu_scalar_index(unsigned int x, unsigned int y){
	return Nx_d*y + x;
}

__device__ __forceinline__ size_t gpu_fieldn_index(unsigned int x, unsigned int y, unsigned int d){
	return (Nx_d*(Ny_d*(d - 1) + y) + x);
}

__global__ void gpu_init_equilibrium(double*, double*, double*, double*, double*);
__global__ void gpu_stream_collide_save(double*, double*, double*, double*, double*, double*, double*, double*, bool);
__global__ void gpu_compute_flow_properties(unsigned int, double*, double*, double*, double*);
__global__ void gpu_print_mesh(int);
__global__ void gpu_initialization(double*, double);

// Poiseulle Flow
__device__ void poiseulle_eval(unsigned int t, unsigned int x, unsigned int y, double *u){

	double delta_p_L = 8*u_max_d*mi_ar_d/(Ny_d*Ny_d);

	double ux = (1/(2*mi_ar_d))*(delta_p_L)*((Ny_d - 1)*y - y*y);

	*u = ux;
}

// Boundary Conditions
__device__ void gpu_bounce_back(unsigned int x, unsigned int y, double *f){
	
	if(y == 0){
		f[gpu_fieldn_index(x, y, 2)] = f[gpu_fieldn_index(x, y, 4)];
		f[gpu_fieldn_index(x, y, 5)] = f[gpu_fieldn_index(x, y, 7)];
		f[gpu_fieldn_index(x, y, 6)] = f[gpu_fieldn_index(x, y, 8)];
	}

	if(y == Ny_d-1){
		f[gpu_fieldn_index(x, y, 4)] = f[gpu_fieldn_index(x, y, 2)];
		f[gpu_fieldn_index(x, y, 7)] = f[gpu_fieldn_index(x, y, 5)];
		f[gpu_fieldn_index(x, y, 8)] = f[gpu_fieldn_index(x, y, 6)];
	}
}

__host__ void init_equilibrium(double *f0, double *f1, double *r, double *u, double *v){

	dim3 grid(Nx/nThreads, Ny, 1);
	dim3 block(nThreads, 1, 1);

	gpu_init_equilibrium<<< grid, block >>>(f0, f1, r, u, v);
	getLastCudaError("gpu_init_equilibrium kernel error");
}

__global__ void gpu_init_equilibrium(double *f0, double *f1, double *r, double *u, double *v){

	unsigned int y = blockIdx.y;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	double rho = r[gpu_scalar_index(x, y)];
	double ux = u[gpu_scalar_index(x, y)];
	double uy = v[gpu_scalar_index(x, y)];

	double A = 1.0/(as_d*as_d);
	double B = 1.0/(2.0*as_d*as_d);

	double w0r = w0_d*rho;
	double wpr = wp_d*rho;
	double wsr = ws_d*rho;
	double omusq = 1.0 - B*(ux*ux + uy*uy);                                                                                                                                              

	double Wrho[] = {w0r, wpr, wpr, wpr, wpr, wsr, wsr, wsr, wsr};

	f0[gpu_field0_index(x, y)] = Wrho[0]*(omusq);
	for(int n = 1; n < q; ++n){
		double eidotu = ux*ex_d[n] + uy*ey_d[n];
		f1[gpu_fieldn_index(x, y, n)] = Wrho[n]*(omusq + A*eidotu*(1.0 + B*eidotu));
	}
}

__host__ void stream_collide_save(double *f0, double *f1, double *f2, double *f0neq, double *f1neq, double *r, double *u, double *v, bool save){

	dim3 grid(Nx/nThreads, Ny, 1);
	dim3 block(nThreads, 1, 1);

	//dim3 grid(1,1,1);
	//dim3 block(1,1,1);

	gpu_stream_collide_save<<< grid, block >>>(f0, f1, f2, f0neq, f1neq, r, u, v, save);
	getLastCudaError("gpu_stream_collide_save kernel error");
}

__global__ void gpu_stream_collide_save(double *f0, double *f1, double *f2, double *f0neq, double *f1neq, double *r, double *u, double *v, bool save){

	const double omega = 1.0/tau_d;

	unsigned int y = blockIdx.y;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	unsigned int xf = (x + 1)%Nx_d;		// Forward
	unsigned int yf = (y + 1)%Ny_d;		// Forward
	unsigned int xb = (Nx_d + x - 1)%Nx_d;	// Backward
	unsigned int yb = (Ny_d + y - 1)%Ny_d; // Backward

	double ft0 = f0[gpu_field0_index(x, y)];

	// Streaming step with Periodic Boundary Conditions
	double ft1 = f1[gpu_fieldn_index(xb, y, 1)];
	double ft2 = f1[gpu_fieldn_index(x, yb, 2)];
	double ft3 = f1[gpu_fieldn_index(xf, y, 3)];
	double ft4 = f1[gpu_fieldn_index(x, yf, 4)];
	double ft5 = f1[gpu_fieldn_index(xb, yb, 5)];
	double ft6 = f1[gpu_fieldn_index(xf, yb, 6)];
	double ft7 = f1[gpu_fieldn_index(xf, yf, 7)];
	double ft8 = f1[gpu_fieldn_index(xb, yf, 8)];

	double f[] = {ft0, ft1, ft2, ft3, ft4, ft5, ft6, ft7, ft8};

	double rho = 0, ux_i = 0, uy_i = 0;

	for(int n = 0; n < q; ++n){
		rho += f[n];
		ux_i += f[n]*ex_d[n];
		uy_i += f[n]*ey_d[n];
	}

	double rhoinv = 1.0/rho;

	double ux = rhoinv*ux_i;
	double uy = rhoinv*uy_i;

	if(save){
		r[gpu_scalar_index(x, y)] = rho;
		u[gpu_scalar_index(x, y)] = ux;
		v[gpu_scalar_index(x, y)] = uy;
	}

	double A = 1.0/(as_d*as_d);
	double B = 1.0/(2.0*as_d*as_d);

	double w0r = w0_d*rho;
	double wpr = wp_d*rho;
	double wsr = ws_d*rho;

	double W[] = {w0_d, wp_d, wp_d, wp_d, wp_d, ws_d, ws_d, ws_d, ws_d};
	double Wrho[] = {w0r, wpr, wpr, wpr, wpr, wsr, wsr, wsr, wsr};

	double omusq = 1.0 - B*(ux*ux + uy*uy);

	// Approximation of fneq
	f0neq[gpu_field0_index(x, y)] = f[0] - Wrho[0]*omusq;
	for(int n = 1; n < q; ++n){
		double eidotu = ux*ex_d[n] + uy*ey_d[n];
		double feq = Wrho[n]*(omusq + A*eidotu*(1.0 + B*eidotu));
		f1neq[gpu_fieldn_index(x, y, n)] = f[n] - feq;
	}

	// Calculating the Viscous stress tensor
	double tauxx = 0, tauxy = 0, tauyy = 0;
	for(int n = 1; n < q; ++n){
		tauxx += f1neq[gpu_fieldn_index(x, y, n)]*ex_d[n]*ex_d[n];
		tauxy += f1neq[gpu_fieldn_index(x, y, n)]*ex_d[n]*ey_d[n];
		tauyy += f1neq[gpu_fieldn_index(x, y, n)]*ey_d[n]*ey_d[n];
	}

	f0[gpu_field0_index(x, y)] = (1.0 - omega)*f0neq[gpu_field0_index(x, y)] + Wrho[0]*(omusq);

	for(int n = 1; n < q; ++n){
		f1neq[gpu_fieldn_index(x, y, n)] = B*W[n]*(tauxx*(A*ex_d[n]*ex_d[n] - 1.0) + 2.0*tauxy*A*ex_d[n]*ey_d[n] + tauyy*(A*ey_d[n]*ey_d[n] - 1.0));
		double eidotu = ux*ex_d[n] + uy*ey_d[n];
		double feq = Wrho[n]*(omusq + A*eidotu*(1.0 + B*eidotu));
		f2[gpu_fieldn_index(x, y, n)] = (1.0 - omega)*f1neq[gpu_fieldn_index(x, y, n)] + feq;
	}

	bool node_solid = solid_d[gpu_scalar_index(x, y)];

	if(node_solid){
		gpu_bounce_back(x, y, f2);
	}

}

__host__ std::vector<double> compute_flow_properties(unsigned int t, double *r, double *u, double *v, std::vector<double> prop, double *prop_gpu, double *prop_host){

	dim3 grid(Nx/nThreads, Ny, 1);
	dim3 block(nThreads, 1, 1);

	gpu_compute_flow_properties<<< grid, block, 3*block.x*sizeof(double) >>>(t, r, u, v, prop_gpu);
	getLastCudaError("gpu_compute_flow_properties kernel error");

	size_t prop_size_bytes = 3*grid.x*grid.y*sizeof(double);
	checkCudaErrors(hipMemcpy(prop_host, prop_gpu, prop_size_bytes, hipMemcpyDeviceToHost));

	double E = 0.0;

	double sumuxe2 = 0.0;
	double sumuxa2 = 0.0;

	for(unsigned int i = 0; i < grid.x*grid.y; ++i){

		E += prop_host[3*i];

		sumuxe2  += prop_host[3*i+1];
		sumuxa2  += prop_host[3*i+2];
	}

	prop.push_back(E);
	prop.push_back(sqrt(sumuxe2/sumuxa2));

	return prop;
}

__global__ void gpu_compute_flow_properties(unsigned int t, double *r, double *u, double *v, double *prop_gpu){

	unsigned int y = blockIdx.y;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	extern __shared__ double data[];

	double *E = data;
    double *uxe2  = data + 1*blockDim.x;
    double *uxa2  = data + 2*blockDim.x;

	double rho = r[gpu_scalar_index(x, y)];
	double ux = u[gpu_scalar_index(x, y)];
	double uy = v[gpu_scalar_index(x, y)];

	E[threadIdx.x] = rho*(ux*ux + uy*uy);

	// compute analytical results
    double uxa;
    poiseulle_eval(t, x, y, &uxa);

     // compute terms for L2 error
    uxe2[threadIdx.x]  = (ux - uxa)*(ux - uxa);
    uxa2[threadIdx.x]  = uxa*uxa;

	__syncthreads();

	if (threadIdx.x == 0){
		
		size_t idx = 3*(gridDim.x*blockIdx.y + blockIdx.x);

		for(int n = 0; n < 3; ++n){
			prop_gpu[idx+n] = 0.0;
		}

		for(int i = 0; i < blockDim.x; ++i){
			prop_gpu[idx  ] += E[i];
            prop_gpu[idx+1] += uxe2[i];
            prop_gpu[idx+2] += uxa2[i];
		}
	}
}

__host__ std::vector<double> report_flow_properties(unsigned int t, double *rho, double *ux, double *uy,
									 double *prop_gpu, double *prop_host, bool msg, bool computeFlowProperties){

	std::vector<double> prop;
	prop = compute_flow_properties(t, rho, ux, uy, prop, prop_gpu, prop_host);

	if(msg){
		if(computeFlowProperties){
			printf("%u, %g, %g\n", t, prop[0], prop[1]);
		}

		if(!quiet){
			printf("Completed timestep %d\n", t);
		}
	}
	
	return prop;
}

__host__ void save_scalar(const std::string name, double *scalar_gpu, double *scalar_host, unsigned int n){

	std::ostringstream path, filename;

	std::string ext = ".dat";

	int ndigits = floor(log10((double)NSTEPS) + 1.0);

	const char* path_results_c = strdup(folder.c_str());

	DIR *dir_results = opendir(path_results_c);
	if(ENOENT == errno){
		mkdir(path_results_c, ACCESSPERMS);
	}

	closedir(dir_results);

	path << folder << name << "/";
	const char* path_c = strdup(path.str().c_str());

	DIR *dir = opendir(path_c);
	if(ENOENT == errno){
		mkdir(path_c, ACCESSPERMS);
	}

	closedir(dir);

	filename << path.str() << name << std::setfill('0') << std::setw(ndigits) << n << ext;
	const char* filename_c = strdup(filename.str().c_str());

	checkCudaErrors(hipMemcpy(scalar_host, scalar_gpu, mem_size_scalar, hipMemcpyDeviceToHost));

	FILE* fout = fopen(filename_c, "wb+");

	fwrite(scalar_host, 1, mem_size_scalar, fout);

	if(ferror(fout)){
		fprintf(stderr, "Error saving to %s\n", filename_c);
		perror("");
	}
	else{
		if(!quiet){
			printf("Saved to %s\n", filename_c);
		}
	}
	fclose(fout);
}

void wrapper_input(unsigned int *nx, unsigned int *ny, double *rho, double *u, double *nu, const double *tau, const double *mi_ar){
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Nx_d), nx, sizeof(unsigned int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Ny_d), ny, sizeof(unsigned int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(rho0_d), rho, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(u_max_d), u, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(nu_d), nu, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(tau_d), tau, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(mi_ar_d), mi_ar, sizeof(double)));
}

void wrapper_lattice(unsigned int *ndir, double *a, double *w_0, double *w_p, double *w_s, double *w_t, double *w_q){
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(q), ndir, sizeof(unsigned int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(as_d), a, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(w0_d), w_0, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(wp_d), w_p, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ws_d), w_s, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(wt_d), w_t, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(wq_d), w_q, sizeof(double)));
}

__host__ int* generate_e(int *e, std::string mode){

	int *temp_e;

	size_t mem_e = ndir*sizeof(int);

	checkCudaErrors(hipMalloc(&temp_e, mem_e));
	checkCudaErrors(hipMemcpy(temp_e, e, mem_e, hipMemcpyHostToDevice));

	if(mode == "x"){
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ex_d), &temp_e, sizeof(temp_e)));
	}
	else if(mode == "y"){
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ey_d), &temp_e, sizeof(temp_e)));
	}

	return temp_e;
}

__host__ bool* generate_mesh(bool *mesh, std::string mode){

	int mode_num;
	bool *temp_mesh;

	checkCudaErrors(hipMalloc(&temp_mesh, mem_mesh));
	checkCudaErrors(hipMemcpy(temp_mesh, mesh, mem_mesh, hipMemcpyHostToDevice));
	

	if(mode == "solid"){
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(solid_d), &temp_mesh, sizeof(temp_mesh)));
		mode_num = 1;
	}

	if(meshprint){
		gpu_print_mesh<<< 1, 1 >>>(mode_num);
		printf("\n");
	}

	return temp_mesh;
}

__global__ void gpu_print_mesh(int mode){
	if(mode == 1){
		for(int y = 0; y < Ny_d; ++y){
			for(int x = 0; x < Nx_d; ++x){
				printf("%d ", solid_d[Nx_d*y + x]);
			}
		printf("\n");
		}
	}
}

__host__ void initialization(double *array, double value){

	dim3 grid(Nx/nThreads, Ny, 1);
	dim3 block(nThreads, 1, 1);

	gpu_initialization<<< grid, block >>>(array, value);
	getLastCudaError("gpu_print_array kernel error");
}

__global__ void gpu_initialization(double *array, double value){

	unsigned int y = blockIdx.y;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	array[gpu_scalar_index(x, y)] = value;
}

__host__ bool* create_pinned_mesh(bool *array){

	bool *pinned;
	const unsigned int bytes = Nx*Ny*sizeof(bool);

	checkCudaErrors(hipHostMalloc((void**)&pinned, bytes));
	memcpy(pinned, array, bytes);
	return pinned;
}

__host__ double* create_pinned_double(){

	double *pinned;

	checkCudaErrors(hipHostMalloc((void**)&pinned, mem_size_scalar));
	return pinned;
}
