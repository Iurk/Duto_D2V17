#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <iomanip>
#include <string>

#define _USE_MATH_DEFINES
#include <math.h>

#include <hip/hip_runtime.h>
#include <dirent.h>
#include <errno.h>
#include <sys/stat.h>

#include "paths.h"
#include "LBM.h"
#include "dados.h"

using namespace myGlobals;

// Input data
__constant__ unsigned int q, Nx_d, Ny_d;
__constant__ double rho0_d, u_max_d, nu_d, tau_d, mi_ar_d;

//Lattice Data
__constant__ double as_d, w0_d, wp_d, ws_d, wt_d, wq_d;
__device__ int *ex_d;
__device__ int *ey_d;

// Mesh data
__device__ bool *solid_d;

__device__ __forceinline__ size_t gpu_scalar_index(unsigned int x, unsigned int y){
	return Nx_d*y + x;
}

__device__ __forceinline__ size_t gpu_fieldn_index(unsigned int x, unsigned int y, unsigned int d){
	return (Nx_d*(Ny_d*(d) + y) + x);
}

__global__ void gpu_init_equilibrium(double*, double*, double*, double*);
__global__ void gpu_stream_collide_save(double*, double*, double*, double*, double*, double*, double*, double*, bool);
__global__ void gpu_compute_flow_properties(unsigned int, double*, double*, double*, double*);
__global__ void gpu_print_mesh(int);
__global__ void gpu_initialization(double*, double);

// Equilibrium
__device__ void gpu_equilibrium(unsigned int x, unsigned int y, double rho, double ux, double uy, double *feq){

	double cs = 1.0/as_d;

	double A = 1.0/(pow(cs, 2));
	double B = 1.0/(2.0*pow(cs, 4));
	double C = 1.0/(6.0*pow(cs, 6));

	double W[] = {w0_d, wp_d, wp_d, wp_d, wp_d, ws_d, ws_d, ws_d, ws_d, wt_d, wt_d, wt_d, wt_d, wq_d, wq_d, wq_d, wq_d};

	for(int n = 0; n < q; ++n){
		double order_1 = A*(ux*ex_d[n] + uy*ey_d[n]);
		double order_2 = B*(pow(ux, 2)*(pow(ex_d[n], 2) - pow(cs, 2)) + 2*ux*uy*ex_d[n]*ey_d[n] + pow(uy, 2)*(pow(ey_d[n], 2) - pow(cs, 2)));
		
		double xxx = pow(ux, 3)*(pow(ex_d[n], 3) - 3*ex_d[n]*pow(cs, 2));
		double yxx = pow(ux, 2)*uy*(pow(ex_d[n], 2)*ey_d[n] - ey_d[n]*pow(cs, 2));
		double xyy = ux*pow(uy, 2)*(ex_d[n]*pow(ey_d[n], 2) - ex_d[n]*pow(cs, 2));
		double yyy = pow(uy, 3)*(pow(ey_d[n], 3) - 3*ey_d[n]*pow(cs, 2));
		double order_3 = C*(xxx + 3*yxx + 3*xyy + yyy);

		feq[gpu_fieldn_index(x, y, n)] = W[n]*rho*(1 + order_1 + order_2 + order_3);
	}
}
// Hermites
__device__ void hermite_polynomial(int ex, int ey, double cs, double *H){

	H[0] = 1;								// 0
	H[1] = ex;								// 1 - x
	H[2] = ey;								// 1 - y
	H[3] = pow(ex, 2) - pow(cs, 2);			// 2 - xx
	H[4] = ex*ey;							// 2 - xy
	H[5] = pow(ey, 2) - pow(cs, 2);			// 2 - yy
	H[6] = pow(ex, 3) - 3*ex*pow(cs, 2);	// 3 - xxx
	H[7] = pow(ex, 2)*ey - ey*pow(cs, 2);	// 3 - yxx
	H[8] = ex*pow(ey, 2) - ex*pow(cs, 2);	// 3 - xyy
	H[9] = pow(ey, 3) - 3*ey*pow(cs, 2);	// 3 - yyy

}

__device__ void hermite_moments(double rho, double ux, double uy, double tauxx, double tauxy, double tauyy, double *a){

	a[0] = rho;											// 0
	a[1] = rho*ux;										// 1 - x
	a[2] = rho*uy;										// 1 - y
	a[3] = rho*pow(ux, 2) + tauxx;						// 2 - xx
	a[4] = rho*ux*uy + tauxy;							// 2 - xy
	a[5] = rho*pow(uy, 2) + tauyy;						// 2 - yy
	a[6] = rho*pow(ux, 3) + 3*ux*tauxx;					// 3 - xxx
	a[7] = rho*pow(ux, 2)*uy + 2*ux*tauxy + uy*tauxx;	// 3 - yxx
	a[8] = rho*ux*pow(uy, 2) + 2*uy*tauxy + ux*tauyy;	// 3 - xyy
	a[9] = rho*pow(uy, 3) + 3*uy*tauyy;					// 3 - yyy
}

// Poiseulle Flow
__device__ void poiseulle_eval(unsigned int t, unsigned int x, unsigned int y, double *u){

	double gradP = -8*u_max_d*mi_ar_d/(pow(Ny_d, 2) - 2*Ny_d);

	double ux = (-1/(2*mi_ar_d))*(gradP)*((Ny_d - 1)*y - pow(y, 2));

	*u = ux;
}

// Boundary Conditions
__device__ void gpu_bounce_back(unsigned int x, unsigned int y, double *f){
	
	if(y == 0){
		f[gpu_fieldn_index(x, y, 2)] = f[gpu_fieldn_index(x, y, 4)];
		f[gpu_fieldn_index(x, y, 5)] = f[gpu_fieldn_index(x, y, 7)];
		f[gpu_fieldn_index(x, y, 6)] = f[gpu_fieldn_index(x, y, 8)];

		//f[gpu_fieldn_index(x, y, 9)] = f[gpu_fieldn_index(x+1, y+1, 11)];
		//f[gpu_fieldn_index(x, y, 10)] = f[gpu_fieldn_index(x-1, y+1, 12)];

		//f[gpu_fieldn_index(x, y, 14)] = f[gpu_fieldn_index(x, y+2, 16)];

		f[gpu_fieldn_index(x+1, y+1, 9)] = f[gpu_fieldn_index(x, y, 11)];
		f[gpu_fieldn_index(x-1, y+1, 10)] = f[gpu_fieldn_index(x, y, 12)];

		f[gpu_fieldn_index(x, y+2, 14)] = f[gpu_fieldn_index(x, y, 16)];
	}

	if(y == Ny_d-1){
		f[gpu_fieldn_index(x, y, 4)] = f[gpu_fieldn_index(x, y, 2)];
		f[gpu_fieldn_index(x, y, 7)] = f[gpu_fieldn_index(x, y, 5)];
		f[gpu_fieldn_index(x, y, 8)] = f[gpu_fieldn_index(x, y, 6)];

		//f[gpu_fieldn_index(x, y, 11)] = f[gpu_fieldn_index(x-1, y-1, 9)];
		//f[gpu_fieldn_index(x, y, 12)] = f[gpu_fieldn_index(x+1, y-1, 10)];

		//f[gpu_fieldn_index(x, y, 16)] = f[gpu_fieldn_index(x, y-2, 14)];

		f[gpu_fieldn_index(x-1, y-1, 11)] = f[gpu_fieldn_index(x, y, 9)];
		f[gpu_fieldn_index(x+1, y-1, 12)] = f[gpu_fieldn_index(x, y, 10)];

		f[gpu_fieldn_index(x, y-2, 16)] = f[gpu_fieldn_index(x, y, 14)];
	}
}

__device__ void gpu_PPBC_inlet(unsigned int x, unsigned int y, double *u, double *v, double *f, double *feq, double *feq_aux){

	double cs = 1.0/as_d;

	// Variables to periodic condition with pressure variation
	double gradP = -8*u_max_d*mi_ar_d/(pow(Ny_d, 2) - 2*Ny_d);
	double gradRho = (Nx_d/(pow(cs, 2)))*gradP;

	double rho_in = rho0_d;
	double rho_out = rho_in + gradRho;

	double ux = u[gpu_scalar_index(Nx_d-1 - x, y)];
	double uy = v[gpu_scalar_index(Nx_d-1 - x, y)];

	for(int n = 0; n < q; ++n){
		gpu_equilibrium(x, y, rho_in, ux, uy, feq_aux);
		f[gpu_fieldn_index(x, y, n)] = feq_aux[n] + (f[gpu_fieldn_index(Nx_d-1 - x, y, n)] - feq[gpu_fieldn_index(Nx_d-1 - x, y, n)]);
	}
}

__device__ void gpu_PPBC_outlet(unsigned int x, unsigned int y, double *u, double *v, double *f, double *feq, double *feq_aux){

	double cs = 1.0/as_d;

	// Variables to periodic condition with pressure variation
	double gradP = -8*u_max_d*mi_ar_d/(pow(Ny_d, 2) - 2*Ny_d);
	double gradRho = (Nx_d/(pow(cs, 2)))*gradP;

	double rho_in = rho0_d;
	double rho_out = rho_in + gradRho;

	double ux = u[gpu_scalar_index(Nx_d-1 - x, y)];
	double uy = v[gpu_scalar_index(Nx_d-1 - x, y)];

	for(int n = 0; n < q; ++n){
			gpu_equilibrium(x, y, rho_out, ux, uy, feq_aux);
			f[gpu_fieldn_index(x, y, n)] = feq_aux[n] + (f[gpu_fieldn_index(Nx_d-1 - x, y, n)] - feq[gpu_fieldn_index(Nx_d-1 - x, y, n)]);	// Periodic with pressure Outlet
		}
}

__host__ void init_equilibrium(double *f1, double *r, double *u, double *v){

	dim3 grid(Nx/nThreads, Ny, 1);
	dim3 block(nThreads, 1, 1);

	gpu_init_equilibrium<<< grid, block >>>(f1, r, u, v);
	getLastCudaError("gpu_init_equilibrium kernel error");
}

__global__ void gpu_init_equilibrium(double *f1, double *r, double *u, double *v){

	unsigned int y = blockIdx.y;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	double rho = r[gpu_scalar_index(x, y)];
	double ux = u[gpu_scalar_index(x, y)];
	double uy = v[gpu_scalar_index(x, y)];

	gpu_equilibrium(x, y, rho, ux, uy, f1);
}

__host__ void stream_collide_save(double *f1, double *f2, double *f1rec, double *feq, double *feq_aux, double *r, double *u, double *v, bool save){

	dim3 grid(Nx/nThreads, Ny, 1);
	dim3 block(nThreads, 1, 1);

	//dim3 grid(1,1,1);
	//dim3 block(1,1,1);

	gpu_stream_collide_save<<< grid, block >>>(f1, f2, f1rec, feq, feq_aux, r, u, v, save);
	getLastCudaError("gpu_stream_collide_save kernel error");
}

__global__ void gpu_stream_collide_save(double *f1, double *f2, double *f1rec, double *feq, double *feq_aux, double *r, double *u, double *v, bool save){

	const double omega = 1.0/tau_d;

	unsigned int y = blockIdx.y;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	unsigned int xf, yf, xb, yb;

	// Streaming Step
	// 1 - 8 directions
	xf = (x + 1)%Nx_d;		// Forward
	yf = (y + 1)%Ny_d;		// Forward
	xb = (Nx_d + x - 1)%Nx_d;	// Backward
	yb = (Ny_d + y - 1)%Ny_d; // Backward

	double ft0 = f1[gpu_fieldn_index(x, y, 0)];
	double ft1 = f1[gpu_fieldn_index(xb, y, 1)];
	double ft2 = f1[gpu_fieldn_index(x, yb, 2)];
	double ft3 = f1[gpu_fieldn_index(xf, y, 3)];
	double ft4 = f1[gpu_fieldn_index(x, yf, 4)];
	double ft5 = f1[gpu_fieldn_index(xb, yb, 5)];
	double ft6 = f1[gpu_fieldn_index(xf, yb, 6)];
	double ft7 = f1[gpu_fieldn_index(xf, yf, 7)];
	double ft8 = f1[gpu_fieldn_index(xb, yf, 8)];

	// 9 - 12 directions
	xf = (x + 2)%Nx_d;		// Forward
	yf = (y + 2)%Ny_d;		// Forward
	xb = (Nx_d + x - 2)%Nx_d;	// Backward
	yb = (Ny_d + y - 2)%Ny_d; // Backward

	double ft9 = f1[gpu_fieldn_index(xb, yb, 9)];
	double ft10 = f1[gpu_fieldn_index(xf, yb, 10)];
	double ft11 = f1[gpu_fieldn_index(xf, yf, 11)];
	double ft12 = f1[gpu_fieldn_index(xb, yf, 12)];

	// 13 - 16 directions
	xf = (x + 3)%Nx_d;		// Forward
	yf = (y + 3)%Ny_d;		// Forward
	xb = (Nx_d + x - 3)%Nx_d;	// Backward
	yb = (Ny_d + y - 3)%Ny_d; // Backward

	double ft13 = f1[gpu_fieldn_index(xb, y, 13)];
	double ft14 = f1[gpu_fieldn_index(x, yb, 14)];
	double ft15 = f1[gpu_fieldn_index(xf, y, 15)];
	double ft16 = f1[gpu_fieldn_index(x, yf, 16)];

	double f[] = {ft0, ft1, ft2, ft3, ft4, ft5, ft6, ft7, ft8, ft9, ft10, ft11, ft12, ft13, ft14, ft15, ft16};
	double rho = 0, ux_i = 0, uy_i = 0, tau_xx = 0, tau_xy = 0, tau_yy = 0;

	for(int n = 0; n < q; ++n){
		rho += f[n];
		ux_i += f[n]*ex_d[n];
		uy_i += f[n]*ey_d[n];
		tau_xx += f[n]*ex_d[n]*ex_d[n];
		tau_xy += f[n]*ex_d[n]*ey_d[n];
		tau_yy += f[n]*ey_d[n]*ey_d[n];
	}

	double ux = ux_i/rho;
	double uy = uy_i/rho;

	if(save){
		r[gpu_scalar_index(x, y)] = rho;
		u[gpu_scalar_index(x, y)] = ux;
		v[gpu_scalar_index(x, y)] = uy;
	}

	double cs = 1.0/as_d;

	double A = 1.0/(pow(cs, 2));
	double B = 1.0/(2.0*pow(cs, 4));
	double C = 1.0/(6.0*pow(cs, 6));

	double W[] = {w0_d, wp_d, wp_d, wp_d, wp_d, ws_d, ws_d, ws_d, ws_d, wt_d, wt_d, wt_d, wt_d, wq_d, wq_d, wq_d, wq_d};

	// Calculating the regularized recursive distribution
	double a[10], H[10];
	for(int n = 0; n < q; ++n){
		hermite_polynomial(ex_d[n], ey_d[n], cs, H);
		hermite_moments(rho, ux, uy, tau_xx, tau_xy, tau_yy, a);

		//					f 			  = W *  (   0      + A*(    x     +     y)     + B*(    xx    +     xy/yx   +    yy)     + C*(   xxx    +    yxx    +    xyy    +    yyy))
		f1rec[gpu_fieldn_index(x, y, n)] = W[n]*(a[0]*H[0] + A*(a[1]*H[1] + a[2]*H[2]) + B*(a[3]*H[3] + 2*a[4]*H[4] + a[5]*H[5]) + C*(a[6]*H[6] + 3*a[7]*H[7] + 3*a[8]*H[8] + a[9]*H[9]));
	}

	// Collision Step
	for(int n = 0; n < q; ++n){
		gpu_equilibrium(x, y, rho, ux, uy, feq);
		f2[gpu_fieldn_index(x, y, n)] = omega*feq[n] + (1 - omega)*f1rec[gpu_fieldn_index(x, y, n)];
	}
/*
	if(x == 0){
		gpu_PPBC_inlet(x, y, u, v, f2, feq, feq_aux);
	}

	if(x == 1){
		gpu_PPBC_inlet(x, y, u, v, f2, feq, feq_aux);
	}

	if(x == 2){
		gpu_PPBC_inlet(x, y, u, v, f2, feq, feq_aux);
	}

	if(x == Nx_d-1){
		gpu_PPBC_outlet(x, y, u, v, f2, feq, feq_aux);
	}

	if(x == Nx_d-2){
		gpu_PPBC_outlet(x, y, u, v, f2, feq, feq_aux);
	}

	if(x == Nx_d-3){
		gpu_PPBC_outlet(x, y, u, v, f2, feq, feq_aux);
	}
*/
	bool node_solid = solid_d[gpu_scalar_index(x, y)];

	// Applying Boundary Conditions
	if(node_solid){
		gpu_bounce_back(x, y, f2);
	}

}

__host__ std::vector<double> compute_flow_properties(unsigned int t, double *r, double *u, double *v, std::vector<double> prop, double *prop_gpu, double *prop_host){

	dim3 grid(Nx/nThreads, Ny, 1);
	dim3 block(nThreads, 1, 1);

	gpu_compute_flow_properties<<< grid, block, 3*block.x*sizeof(double) >>>(t, r, u, v, prop_gpu);
	getLastCudaError("gpu_compute_flow_properties kernel error");

	size_t prop_size_bytes = 3*grid.x*grid.y*sizeof(double);
	checkCudaErrors(hipMemcpy(prop_host, prop_gpu, prop_size_bytes, hipMemcpyDeviceToHost));

	double E = 0.0;

	double sumuxe2 = 0.0;
	double sumuxa2 = 0.0;

	for(unsigned int i = 0; i < grid.x*grid.y; ++i){

		E += prop_host[3*i];

		sumuxe2  += prop_host[3*i+1];
		sumuxa2  += prop_host[3*i+2];
	}

	prop.push_back(E);
	prop.push_back(sqrt(sumuxe2/sumuxa2));

	return prop;
}

__global__ void gpu_compute_flow_properties(unsigned int t, double *r, double *u, double *v, double *prop_gpu){

	unsigned int y = blockIdx.y;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	extern __shared__ double data[];

	double *E = data;
    double *uxe2  = data + 1*blockDim.x;
    double *uxa2  = data + 2*blockDim.x;

	double rho = r[gpu_scalar_index(x, y)];
	double ux = u[gpu_scalar_index(x, y)];
	double uy = v[gpu_scalar_index(x, y)];

	E[threadIdx.x] = rho*(ux*ux + uy*uy);

	// compute analytical results
    double uxa;
    poiseulle_eval(t, x, y, &uxa);

     // compute terms for L2 error
    uxe2[threadIdx.x]  = (ux - uxa)*(ux - uxa);
    uxa2[threadIdx.x]  = uxa*uxa;

	__syncthreads();

	if (threadIdx.x == 0){
		
		size_t idx = 3*(gridDim.x*blockIdx.y + blockIdx.x);

		for(int n = 0; n < 3; ++n){
			prop_gpu[idx+n] = 0.0;
		}

		for(int i = 0; i < blockDim.x; ++i){
			prop_gpu[idx  ] += E[i];
            prop_gpu[idx+1] += uxe2[i];
            prop_gpu[idx+2] += uxa2[i];
		}
	}
}

__host__ std::vector<double> report_flow_properties(unsigned int t, double *rho, double *ux, double *uy,
									 double *prop_gpu, double *prop_host, bool msg, bool computeFlowProperties){

	std::vector<double> prop;
	prop = compute_flow_properties(t, rho, ux, uy, prop, prop_gpu, prop_host);

	if(msg){
		if(computeFlowProperties){
			printf("%u, %g, %g\n", t, prop[0], prop[1]);
		}

		if(!quiet){
			printf("Completed timestep %d\n", t);
		}
	}
	
	return prop;
}

__host__ void save_scalar(const std::string name, double *scalar_gpu, double *scalar_host, unsigned int n){

	std::ostringstream path, filename;

	std::string ext = ".dat";

	int ndigits = floor(log10((double)NSTEPS) + 1.0);

	const char* path_results_c = strdup(folder.c_str());

	DIR *dir_results = opendir(path_results_c);
	if(ENOENT == errno){
		mkdir(path_results_c, ACCESSPERMS);
	}

	closedir(dir_results);

	path << folder << name << "/";
	const char* path_c = strdup(path.str().c_str());

	DIR *dir = opendir(path_c);
	if(ENOENT == errno){
		mkdir(path_c, ACCESSPERMS);
	}

	closedir(dir);

	filename << path.str() << name << std::setfill('0') << std::setw(ndigits) << n << ext;
	const char* filename_c = strdup(filename.str().c_str());

	checkCudaErrors(hipMemcpy(scalar_host, scalar_gpu, mem_size_scalar, hipMemcpyDeviceToHost));

	FILE* fout = fopen(filename_c, "wb+");

	fwrite(scalar_host, 1, mem_size_scalar, fout);

	if(ferror(fout)){
		fprintf(stderr, "Error saving to %s\n", filename_c);
		perror("");
	}
	else{
		if(!quiet){
			printf("Saved to %s\n", filename_c);
		}
	}
	fclose(fout);
}

void wrapper_input(unsigned int *nx, unsigned int *ny, double *rho, double *u, double *nu, const double *tau, const double *mi_ar){
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Nx_d), nx, sizeof(unsigned int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(Ny_d), ny, sizeof(unsigned int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(rho0_d), rho, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(u_max_d), u, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(nu_d), nu, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(tau_d), tau, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(mi_ar_d), mi_ar, sizeof(double)));
}

void wrapper_lattice(unsigned int *ndir, double *a, double *w_0, double *w_p, double *w_s, double *w_t, double *w_q){
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(q), ndir, sizeof(unsigned int)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(as_d), a, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(w0_d), w_0, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(wp_d), w_p, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ws_d), w_s, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(wt_d), w_t, sizeof(double)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(wq_d), w_q, sizeof(double)));
}

__host__ int* generate_e(int *e, std::string mode){

	int *temp_e;

	size_t mem_e = ndir*sizeof(int);

	checkCudaErrors(hipMalloc(&temp_e, mem_e));
	checkCudaErrors(hipMemcpy(temp_e, e, mem_e, hipMemcpyHostToDevice));

	if(mode == "x"){
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ex_d), &temp_e, sizeof(temp_e)));
	}
	else if(mode == "y"){
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(ey_d), &temp_e, sizeof(temp_e)));
	}

	return temp_e;
}

__host__ bool* generate_mesh(bool *mesh, std::string mode){

	int mode_num;
	bool *temp_mesh;

	checkCudaErrors(hipMalloc(&temp_mesh, mem_mesh));
	checkCudaErrors(hipMemcpy(temp_mesh, mesh, mem_mesh, hipMemcpyHostToDevice));
	

	if(mode == "solid"){
		checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(solid_d), &temp_mesh, sizeof(temp_mesh)));
		mode_num = 1;
	}

	if(meshprint){
		gpu_print_mesh<<< 1, 1 >>>(mode_num);
		printf("\n");
	}

	return temp_mesh;
}

__global__ void gpu_print_mesh(int mode){
	if(mode == 1){
		for(int y = 0; y < Ny_d; ++y){
			for(int x = 0; x < Nx_d; ++x){
				printf("%d ", solid_d[Nx_d*y + x]);
			}
		printf("\n");
		}
	}
}

__host__ void initialization(double *array, double value){

	dim3 grid(Nx/nThreads, Ny, 1);
	dim3 block(nThreads, 1, 1);

	gpu_initialization<<< grid, block >>>(array, value);
	getLastCudaError("gpu_print_array kernel error");
}

__global__ void gpu_initialization(double *array, double value){

	unsigned int y = blockIdx.y;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	array[gpu_scalar_index(x, y)] = value;
}

__host__ bool* create_pinned_mesh(bool *array){

	bool *pinned;
	const unsigned int bytes = Nx*Ny*sizeof(bool);

	checkCudaErrors(hipHostMalloc((void**)&pinned, bytes));
	memcpy(pinned, array, bytes);
	return pinned;
}

__host__ double* create_pinned_double(){

	double *pinned;

	checkCudaErrors(hipHostMalloc((void**)&pinned, mem_size_scalar));
	return pinned;
}
