#include <stdio.h>
#include <stdlib.h>
#include <iomanip>

#include "seconds.h"
#include "LBM.h"
#include "dados.h"

using namespace myGlobals;

int main(int argc, char const *argv[]){

	// Writing Simulation Parameters
	printf("Simulating the Poiseulle Flow\n");
	printf("       Domain size: %ux%u\n", Nx, Ny);
	printf("                nu: %g\n", nu);
	printf("               tau: %g\n", tau);
	printf("             u_max: %g\n", u_max);
	printf("              rho0: %g\n", rho0);
	printf("                Re: %g\n", Re);
	printf("       Times Stpes: %u\n", NSTEPS);
	printf("        Save every: %u\n", NSAVE);
	printf("     Message every: %u\n", NMSG);
	printf("Velocity Tolerance: %g\n", erro_max);
	printf("\n");

	double bytesPerMiB = 1024.0*1024.0;
	double bytesPerGiB = 1024.0*1024.0*1024.0;

	// Getting Device Info and Writing then
	checkCudaErrors(hipSetDevice(0));
	int deviceId = 0;
	checkCudaErrors(hipGetDevice(&deviceId));

	hipDeviceProp_t deviceProp;
	checkCudaErrors(hipGetDeviceProperties(&deviceProp, deviceId));

	size_t gpu_free_mem, gpu_total_mem;
	checkCudaErrors(hipMemGetInfo(&gpu_free_mem, &gpu_total_mem));

	printf("CUDA information\n");
	printf("      Using device: %d\n", deviceId);
	printf("              Name: %s\n", deviceProp.name);
	printf("   Multiprocessors: %d\n", deviceProp.multiProcessorCount);
	printf("Compute Capability: %d.%d\n", deviceProp.major, deviceProp.minor);
	printf("     Global Memory: %.1f MiB\n", deviceProp.totalGlobalMem/bytesPerMiB);
	printf("       Free Memory: %.1f MiB\n", gpu_free_mem/bytesPerMiB);
	printf("\n");

	// Declaration and Allocation in device Memory
	double *f1_gpu, *f2_gpu, *f1rec_gpu, *F_gpu;
	double *rho_gpu, *ux_gpu, *uy_gpu, *ux_old_gpu;
	double *prop_gpu, *conv_gpu;

	checkCudaErrors(hipMalloc((void**)&f1_gpu, mem_size_ndir));
	checkCudaErrors(hipMalloc((void**)&f2_gpu, mem_size_ndir));
	checkCudaErrors(hipMalloc((void**)&f1rec_gpu, mem_size_ndir));
	checkCudaErrors(hipMalloc((void**)&F_gpu, mem_size_ndir));
	checkCudaErrors(hipMalloc((void**)&rho_gpu, mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&ux_gpu, mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&uy_gpu, mem_size_scalar));
	checkCudaErrors(hipMalloc((void**)&ux_old_gpu, mem_size_scalar));

	const size_t mem_size_conv = 2*Nx/nThreads*Ny*sizeof(double);
	const size_t mem_size_props = 3*Nx/nThreads*Ny*sizeof(double);
	checkCudaErrors(hipMalloc((void**)&prop_gpu, mem_size_props));
	checkCudaErrors(hipMalloc((void**)&conv_gpu, mem_size_conv));

	double *scalar_host, *conv_host;
	scalar_host = create_pinned_double();
	conv_host = create_pinned_double();
	if(scalar_host == NULL){
		fprintf(stderr, "Error: unable to allocate required memory (%.1f MiB).\n", mem_size_scalar/bytesPerMiB);
		exit(-1);
	}

	size_t total_mem_bytes = 4*mem_size_ndir + 3*mem_size_scalar + mem_size_props;
	
	// Creating Events for time measure
	hipEvent_t start, stop;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));

	// Allocation of Input data in Device constant memory
	wrapper_input(&Nx, &Ny, &rho0, &u_max, &nu, &tau, &mi_ar);

	// Allocation of Lattice data in Device constant and global memory
	wrapper_lattice(&ndir, &as, &w0, &wp, &ws, &wt, &wq);

	int *ex_gpu, *ey_gpu;

	ex_gpu = generate_e(ex, "x");
	ey_gpu = generate_e(ey, "y");

	bool *walls_p, *inlet_p, *outlet_p;
	bool *walls_gpu, *inlet_gpu, *outlet_gpu;

	walls_p = create_pinned_mesh(walls);
	inlet_p = create_pinned_mesh(inlet);
	outlet_p = create_pinned_mesh(outlet);

	// Generating Mesh
	walls_gpu = generate_mesh(walls_p, "walls");
	inlet_gpu = generate_mesh(inlet_p, "inlet");
	outlet_gpu = generate_mesh(outlet_p, "outlet");

	// Initialization
	initialization(rho_gpu, rho0);
	initialization(ux_gpu, u_max);
	initialization(uy_gpu, 0.0);
	initialization(ux_old_gpu, 0.0);

	init_equilibrium(f1_gpu, rho_gpu, ux_gpu, uy_gpu);
	checkCudaErrors(hipMemset(f1rec_gpu, 0, mem_size_ndir));

	save_scalar("rho",rho_gpu, scalar_host, 0);
	save_scalar("ux", ux_gpu, scalar_host, 0);
	save_scalar("uy", uy_gpu, scalar_host, 0);
	
	// Simulation Start
	double begin = seconds();
	checkCudaErrors(hipEventRecord(start, 0));

	double conv_error;

	// Main Loop
	printf("Starting main loop...\n");
	std::cout << std::setw(10) << "Timestep" << std::setw(10) << "E" << std::setw(10) << "L2" << std::setw(20) << "Convergence" << std::endl;
	for(unsigned int n = 0; n < NSTEPS; ++n){
		bool save = (n+1)%NSAVE == 0;
		bool msg = (n+1)%NMSG == 0;
		bool need_scalars = save || (msg && computeFlowProperties);
/*
		double *ux_test;

		ux_test = (double*)malloc(mem_size_scalar);
		checkCudaErrors(hipMemcpy(ux_test, ux_gpu, mem_size_scalar, hipMemcpyDeviceToHost));

		for(int y = 0; y < Ny; ++y){
			std::cout << y << "-> ";
			for(int x = 0; x < Nx; ++x){
				std::cout << ux_test[Nx*y+x] << " ";
			}
			std::cout << std::endl;
		}
*/
		stream_collide_save(f1_gpu, f2_gpu, f1rec_gpu, F_gpu, rho_gpu, ux_gpu, uy_gpu, need_scalars);

		if(save){
			save_scalar("rho",rho_gpu, scalar_host, n+1);
			save_scalar("ux", ux_gpu, scalar_host, n+1);
			save_scalar("uy", uy_gpu, scalar_host, n+1);
		}

		double *temp = f1_gpu;
		f1_gpu = f2_gpu;
		f2_gpu = temp;
		
		conv_error = compute_convergence(ux_gpu, ux_old_gpu, conv_gpu, conv_host);
		report_flow_properties(n+1, conv_error, rho_gpu, ux_gpu, uy_gpu, prop_gpu, scalar_host, msg, computeFlowProperties);

		if(conv_error < erro_max){
			msg = 0 == 0;
			report_flow_properties(n+1, conv_error, rho_gpu, ux_gpu, uy_gpu, prop_gpu, scalar_host, msg, computeFlowProperties);
			break;
		}

		checkCudaErrors(hipMemcpy(ux_old_gpu, ux_gpu, mem_size_scalar, hipMemcpyDeviceToDevice));
	}

	// Measuring time
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	float miliseconds = 0.0f;
	checkCudaErrors(hipEventElapsedTime(&miliseconds, start, stop));

	double end = seconds();
	double runtime = end - begin;
	double gpu_runtime = 0.001*miliseconds;

	size_t doubles_read = ndir;
	size_t doubles_wirtten = ndir;
	size_t doubles_saved = 3;

	size_t nodes_updated = NSTEPS*size_t(Nx*Ny);
	size_t nodes_saved = (NSTEPS/NSAVE)*size_t(Nx*Ny);
	double speed = nodes_updated/(1e6*runtime);

	double bandwidth = (nodes_updated*(doubles_read + doubles_wirtten) + nodes_saved*(doubles_saved))*sizeof(double)/(runtime*bytesPerGiB);

	// Writing the performance
	printf("Performance Information\n");
	printf(" Memory Allocated (GPU): %.1f (MiB)\n", total_mem_bytes/bytesPerMiB);
	printf("Memory Allocated (host): %.1f (MiB)\n", mem_size_scalar/bytesPerMiB);
	printf("              Timesteps: %u\n", NSTEPS);
	printf("             Clock Time: %.3f (s)\n", runtime);
	printf("            GPU runtime: %.3f (s)\n", gpu_runtime);
	printf("                  Speed: %.2f (Mlups)\n", speed);
	printf("               Bandwith: %.1f (GiB/s)\n", bandwidth);

	// Cleaning up

	// Destroying Events
	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));

	// Freeing Device and CPU Memory
	// LBM variables
	checkCudaErrors(hipFree(f1_gpu));
	checkCudaErrors(hipFree(f2_gpu));
	checkCudaErrors(hipFree(f1rec_gpu));
	checkCudaErrors(hipFree(F_gpu));
	checkCudaErrors(hipFree(rho_gpu));
	checkCudaErrors(hipFree(ux_gpu));
	checkCudaErrors(hipFree(uy_gpu));
	checkCudaErrors(hipFree(ux_old_gpu));
	checkCudaErrors(hipFree(prop_gpu));
	checkCudaErrors(hipFree(conv_gpu));
	checkCudaErrors(hipFree(ex_gpu));
	checkCudaErrors(hipFree(ey_gpu));

	// Mesh arrays
	checkCudaErrors(hipFree(walls_gpu));
	checkCudaErrors(hipHostFree(walls_p));
	checkCudaErrors(hipFree(inlet_gpu));
	checkCudaErrors(hipHostFree(inlet_p));
	checkCudaErrors(hipFree(outlet_gpu));
	checkCudaErrors(hipHostFree(outlet_p));

	// Host arrays
	checkCudaErrors(hipHostFree(scalar_host));

	hipDeviceReset();

	return 0;
}
