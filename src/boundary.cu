#include "hip/hip_runtime.h"
#include "LBM.h"
#include "dados.h"
#include "boundary.h"

using namespace myGlobals;

__device__ __forceinline__ size_t gpu_scalar_index(unsigned int x, unsigned int y){
	return Nx_d*y + x;
}

__device__ __forceinline__ size_t gpu_fieldn_index(unsigned int x, unsigned int y, unsigned int d){
	return (Nx_d*(Ny_d*(d) + y) + x);
}

__global__ void gpu_inlet(double, double*, double*, double*, double*, double*, double*, double*);
__global__ void gpu_bounce_back(double*);
__global__ void gpu_outlet(double, double*);

// Boundary Conditions
__device__ void device_bounce_back(unsigned int x, unsigned int y, double *f){
	
	if(y == 0){
		f[gpu_fieldn_index(x, y, 2)] = f[gpu_fieldn_index(x, y, 4)];
		f[gpu_fieldn_index(x, y, 5)] = f[gpu_fieldn_index(x, y, 7)];
		f[gpu_fieldn_index(x, y, 6)] = f[gpu_fieldn_index(x, y, 8)];

		f[gpu_fieldn_index(x, y, 9)] = f[gpu_fieldn_index(x+1, y+1, 11)];
		f[gpu_fieldn_index(x, y, 10)] = f[gpu_fieldn_index(x-1, y+1, 12)];

		f[gpu_fieldn_index(x, y, 14)] = f[gpu_fieldn_index(x, y+2, 16)];
	}
	else if(y == 1){
		f[gpu_fieldn_index(x, y, 9)] = f[gpu_fieldn_index(x-1, y-1, 11)];
		f[gpu_fieldn_index(x, y, 10)] = f[gpu_fieldn_index(x+1, y-1, 12)];

		f[gpu_fieldn_index(x, y, 14)] = f[gpu_fieldn_index(x, y, 16)];
	}
	else if(y == 2){
		f[gpu_fieldn_index(x, y, 14)] = f[gpu_fieldn_index(x, y-2, 16)];
	}

	if(y == Ny_d-1){
		f[gpu_fieldn_index(x, y, 4)] = f[gpu_fieldn_index(x, y, 2)];
		f[gpu_fieldn_index(x, y, 7)] = f[gpu_fieldn_index(x, y, 5)];
		f[gpu_fieldn_index(x, y, 8)] = f[gpu_fieldn_index(x, y, 6)];

		f[gpu_fieldn_index(x, y, 11)] = f[gpu_fieldn_index(x-1, y-1, 9)];
		f[gpu_fieldn_index(x, y, 12)] = f[gpu_fieldn_index(x+1, y-1, 10)];

		f[gpu_fieldn_index(x, y, 16)] = f[gpu_fieldn_index(x, y-2, 14)];
	}
	else if(y == Ny_d-2){
		f[gpu_fieldn_index(x, y, 11)] = f[gpu_fieldn_index(x+1, y+1, 9)];
		f[gpu_fieldn_index(x, y, 12)] = f[gpu_fieldn_index(x-1, y+1, 10)];

		f[gpu_fieldn_index(x, y, 16)] = f[gpu_fieldn_index(x, y, 14)];
	}
	else if(y == Ny_d-3){
		f[gpu_fieldn_index(x, y, 16)] = f[gpu_fieldn_index(x, y+2, 14)];
	}
}

__device__ void device_inlet(unsigned int x, unsigned int y, double ux_in, double *f, double *feq, double *frec, double *r, double *u, double *v, double *txy){

	double uy_in = 0.0;

	double rho, tauxy;

	if(x == 0){
		unsigned int NI = 11;
		unsigned int I[11] = {0, 2, 3, 4, 6, 7, 10, 11, 14, 15, 16};

		double rhoI = 0.0, rhoaxy = 0.0;
		for(int n = 0; n < NI; ++n){
			unsigned int ni = I[n];
			rhoI += f[gpu_fieldn_index(x, y, ni)];
			rhoaxy += f[gpu_fieldn_index(x, y, ni)]*ex_d[ni]*ey_d[ni];
		}

		double ux_in2 = ux_in*ux_in;
		double ux_in3 = ux_in*ux_in*ux_in;

		rho = (129600*rhoI)/((5*sqrt(193.0)+5525)*ux_in3 + (-31380-1740*sqrt(193.0))*ux_in2 + (-54144-720.0*sqrt(193.0))*ux_in + (808*sqrt(193.0)+94712));
		tauxy = (-270*rhoaxy)/((4*sqrt(193.0)+190)*ux_in - 135);

		r[gpu_scalar_index(x, y)] = rho;
		u[gpu_scalar_index(x, y)] = ux_in;
		v[gpu_scalar_index(x, y)] = uy_in;
		txy[gpu_scalar_index(x, y)] = tauxy;

		gpu_recursive(x, y, rho, ux_in, uy_in, 0.0, tauxy, 0.0, frec);

		for(int n = 0; n < q; ++n){
			f[gpu_fieldn_index(x, y, n)] = frec[gpu_fieldn_index(x, y, n)];
		}
	}

	else if(x == 1){
		f[gpu_fieldn_index(x, y, 9)] = f[gpu_fieldn_index(x, y, 11)] - feq[gpu_fieldn_index(x, y, 11)] + feq[gpu_fieldn_index(x, y, 9)];
		f[gpu_fieldn_index(x, y, 12)] = f[gpu_fieldn_index(x, y, 10)] - feq[gpu_fieldn_index(x, y, 10)] + feq[gpu_fieldn_index(x, y, 12)];

		if(y > 0 && y < Ny_d-2){
			f[gpu_fieldn_index(x, y, 9)] = (1.0/3.0)*f[gpu_fieldn_index(x-1, y-1, 9)] + f[gpu_fieldn_index(x+1, y+1, 9)] - (1.0/3.0)*f[gpu_fieldn_index(x+2, y+2, 9)];
		}

		if(y > 1 && y < Ny_d-1){
			f[gpu_fieldn_index(x, y, 12)] = (1.0/3.0)*f[gpu_fieldn_index(x-1, y+1, 12)] + f[gpu_fieldn_index(x+1, y-1, 12)] - (1.0/3.0)*f[gpu_fieldn_index(x+2, y-2, 12)];
		}
		
		f[gpu_fieldn_index(x, y, 13)] = (1.0/3.0)*f[gpu_fieldn_index(x-1, y, 13)] + f[gpu_fieldn_index(x+1, y, 13)] - (1.0/3.0)*f[gpu_fieldn_index(x+2, y, 13)];
	}

	else if(x == 2){
		f[gpu_fieldn_index(x, y, 13)] = (1.0/6.0)*f[gpu_fieldn_index(x-2, y, 13)] + (4.0/3.0)*f[gpu_fieldn_index(x+1, y, 13)] - (1.0/2.0)*f[gpu_fieldn_index(x+2, y, 13)];
	}
}

__host__ void bounce_back(double *f){

	dim3 grid(Nx/nThreads, Ny, 1);
	dim3 block(nThreads, 1, 1);

	gpu_bounce_back<<< grid, block >>>(f);
	getLastCudaError("gpu_bounce_back kernel error");
}

__global__ void gpu_bounce_back(double *f){

	unsigned int y = blockIdx.y;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	bool node_walls = walls_d[gpu_scalar_index(x, y)];
	if(node_walls){
		device_bounce_back(x, y, f);
	}
}

__host__ void inlet_BC(double ux_in, double *f, double *feq, double *frec, double *r, double *u, double *v, double *txy){

	dim3 grid(Nx/nThreads, Ny, 1);
	dim3 block(nThreads, 1, 1);

	gpu_inlet<<< grid, block >>>(ux_in, f, feq, frec, r, u, v, txy);
	getLastCudaError("gpu_inlet kernel error");
}

__global__ void gpu_inlet(double ux_in, double *f, double *feq, double *frec, double *r, double *u, double *v, double *txy){

	unsigned int y = blockIdx.y;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	bool node_inlet = inlet_d[gpu_scalar_index(x, y)];
	if(node_inlet){
		device_inlet(x, y, ux_in, f, feq, frec, r, u, v, txy);
	}
}

__host__ void outlet_BC(double rho, double *f){

	dim3 grid(Nx/nThreads, Ny, 1);
	dim3 block(nThreads, 1, 1);

	gpu_outlet<<< grid, block >>>(rho, f);
	getLastCudaError("gpu_outlet kernel error");
}

__global__ void gpu_outlet(double rho, double *f){

	unsigned int y = blockIdx.y;
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

	bool node_outlet = outlet_d[gpu_scalar_index(x, y)];
	if(node_outlet){

		double sumRho = 0.0;
		for(int n = 0; n < q; ++n){
			sumRho += f[gpu_fieldn_index(x-3, y, n)];
		}

		for(int n = 0; n < q; ++n){
			f[gpu_fieldn_index(x, y, n)] = (rho/sumRho)*f[gpu_fieldn_index(x-3, y, n)];
		}
	}
}